#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void naiveMatmulKernel(const float *A, const float *B, float *C,
                                  int m, int n, int k) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int col = bx * blockDim.x + tx;
  int row = by * blockDim.y + ty;

  if ((col < n) && (row < m)) {
    float sum = 0.f;
    for (int i = 0; i < k; ++i) {
      sum += (A[row * k + i] * B[i * n + col]);
    }
    C[row * n + col] = sum;
  }
}

__global__ void tiledMatmulKernel(const float *A, const float *B, float *C,
                                  int m, int n, int k) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int col = bx * blockDim.x + tx;
  int row = by * blockDim.y + ty;

  int num_stages = (k + TILE_SIZE - 1) / TILE_SIZE;

  __shared__ float smem_A[TILE_SIZE][TILE_SIZE];
  __shared__ float smem_B[TILE_SIZE][TILE_SIZE];

  float sum = 0.f;
  for (int stage = 0; stage < num_stages; ++stage) {
    int k_offset = stage * TILE_SIZE;

    if (row < m && tx + k_offset < k) {
      smem_A[ty][tx] = A[row * k + tx + k_offset];
    }
    if (col < n && ty + k_offset < k) {
      smem_B[ty][tx] = B[(ty + k_offset) * n + col];
    }
    __syncthreads();

#pragma unroll
    for (int i = 0; i < TILE_SIZE; ++i) {
      if (k_offset + i < k) {
        sum += smem_A[ty][i] * smem_B[i][tx];
      }
    }
  }
  if ((col < n) && (row < m)) {
    C[row * n + col] = sum;
  }
}

int main() {
  int m = 33;
  int n = 129;
  int k = 257;

  float *A_h = (float *)malloc(m * k * sizeof(float));
  float *B_h = (float *)malloc(k * n * sizeof(float));
  float *C_h = (float *)malloc(m * n * sizeof(float));
  float *C_h_naive = (float *)malloc(m * n * sizeof(float));

  for (int i = 0; i < m * k; i++) {
    A_h[i] = rand() / (float)RAND_MAX;
  }
  for (int i = 0; i < k * n; i++) {
    B_h[i] = rand() / (float)RAND_MAX;
  }

  float *A_d, *B_d, *C_d, *C_d_naive;
  hipMalloc(&A_d, m * k * sizeof(float));
  hipMalloc(&B_d, k * n * sizeof(float));
  hipMalloc(&C_d, m * n * sizeof(float));
  hipMalloc(&C_d_naive, m * n * sizeof(float));

  hipMemcpy(A_d, A_h, m * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, k * n * sizeof(float), hipMemcpyHostToDevice);

  dim3 block_size(TILE_SIZE, TILE_SIZE);
  dim3 grid_size((n + TILE_SIZE - 1) / TILE_SIZE,
                 (m + TILE_SIZE - 1) / TILE_SIZE);

  tiledMatmulKernel<<<grid_size, block_size>>>(A_d, B_d, C_d, m, n, k);
  hipMemcpy(C_h, C_d, m * n * sizeof(float), hipMemcpyDeviceToHost);
  printf("Tiled matmul done\n");
  naiveMatmulKernel<<<grid_size, block_size>>>(A_d, B_d, C_d_naive, m, n, k);
  hipMemcpy(C_h_naive, C_d_naive, m * n * sizeof(float),
             hipMemcpyDeviceToHost);
  printf("Naive matmul done\n");

  for (int i = 0; i < m * n; i++) {
    if (C_h[i] != C_h_naive[i]) {
      printf("Error at %d: %f != %f\n", i, C_h[i], C_h_naive[i]);
    }
  }

  free(A_h);
  free(B_h);
  free(C_h);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}
